#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define NUM_VARIATIONS 4
#define NUM_XFORMS 3

struct affine {
  float a;
  float b;
  float c;
  float d;
  float e;
  float f;
};

struct xform {
  float weights[NUM_VARIATIONS];
  float weight;
  affine pre_affine;
  affine post_affine;
  float r;
  float g;
  float b;
  float a;
};

struct pixel {
  char r;
  char g;
  char b;
};

struct particle {
  float x;
  float y;
};

struct hist {
  unsigned int count;
  float r;
  float g;
  float b;
  float a;
};

__device__ void var0_linear(const particle &in, particle *out, float weight) {
  out->x += in.x * weight;
  out->y += in.y * weight;
}

__device__ void var1_sinusoidal(const particle &in, particle *out,
                                float weight) {
  out->x += weight * sin(in.x);
  out->y += weight * sin(in.y);
}

__device__ void var2_spherical(const particle &in, particle *out,
                               float weight) {
  const float eps = 0.000001f;
  float r2 = weight / (in.x * in.x + in.y * in.y + eps);
  out->x += r2 * in.x;
  out->y += r2 * in.y;
}

__device__ void var3_swirl(const particle &in, particle *out, float weight) {
  float sumsq = (in.x * in.x + in.y * in.y);

  float c1 = sin(sumsq);
  float c2 = cos(sumsq);
  float nx = c1 * in.x - c2 * in.y;
  float ny = c2 * in.x + c1 * in.y;

  out->x += weight * nx;
  out->y += weight * ny;
}

/*
__device__ void var4_horseshoe(flam3_iter_helper *f, double weight) {
double r = weight / (f->precalc_sqrt + EPS);

f->p0 += (f->tx - f->ty) * (f->tx + f->ty) * r;
f->p1 += 2.0 * f->tx * f->ty * r;
}
*/

__device__ particle apply_affine(const particle &in, const affine &a) {
  return {in.x * a.a + in.y * a.b + a.c, in.x * a.d + in.y * a.e + a.f};
}

__global__ void sheep(int num_sheep, int iterations, hist *histogram, int w,
                      int h, xform *xforms) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  hiprandState random_state;
  hiprandState global_random_state;
  hiprand_init(index, 0, 0, &random_state);
  hiprand_init(0, 0, 0, &global_random_state);
  const int wait_time = 200;

  float total_weight = 0.0f;
  float weight_boundaries[NUM_XFORMS] = {0.};
  for (int i = 0; i < NUM_XFORMS; ++i) {
    total_weight += xforms[i].weight;
    weight_boundaries[i] = total_weight;
  }

  // Random sampling.
  particle state = {hiprand_uniform(&random_state) * 2.0 - 1.0,
                    hiprand_uniform(&random_state) * 2.0 - 1.0};
  state.x = 0;
  state.y = 0;

  // same starting point
  for (int iter = 0; iter < iterations && index < num_sheep; ++iter) {
    float xform_sample = hiprand_uniform(&random_state) * total_weight;

    int xform_index = 0;
    for (xform_index = 0; xform_index < NUM_XFORMS - 1; ++xform_index) {
      if (xform_sample < weight_boundaries[xform_index]) {
        break;
      }
    }

    const xform &xform = xforms[xform_index];

    particle input = apply_affine(state, xform.pre_affine);
    particle next = {0., 0.};

    var0_linear(input, &next, xform.weights[0]);
    var1_sinusoidal(input, &next, xform.weights[1]);
    var2_spherical(input, &next, xform.weights[2]);
    var3_swirl(input, &next, xform.weights[3]);

    next = apply_affine(next, xform.post_affine);

    state = next;

    // TODO: Does atomicAdd prevent race condition?
    if (iter > wait_time) {
      int ix = (int)((state.x + 1.0) * (w / 2));
      int iy = (int)((state.y + 1.0) * (h / 2));
      if (ix >= 0 && ix < w && iy >= 0 && iy < h) {
        atomicInc(&(histogram[iy * w + ix].count), 1);

        // race condition for color..., lets just try it
        histogram[iy * w + ix].r = (xform.r + histogram[iy * w + ix].r) / 2.0f;
        histogram[iy * w + ix].g = (xform.g + histogram[iy * w + ix].g) / 2.0f;
        histogram[iy * w + ix].b = (xform.b + histogram[iy * w + ix].b) / 2.0f;
        histogram[iy * w + ix].a = (xform.a + histogram[iy * w + ix].a) / 2.0f;
      }
    }
  }
}

__global__ void histogram_to_image(hist *histogram, pixel *pixels, int w,
                                   int h) {
  const int index = threadIdx.x;
  const int stride = blockDim.x;
  for (int i = index; i < w * h; i += stride) {
    pixels[i].r = 0;
    pixels[i].g = 0;
    pixels[i].b = 0;

    // we have an average
    // lets map to ( val - average) / avarege +
    if (histogram[i].count > 0) {
      const float p = powf(histogram[i].a, 1.0f / 2.2f);
      pixels[i].r = histogram[i].r * p * 255;
      pixels[i].g = histogram[i].g * p * 255;
      pixels[i].b = histogram[i].b * p * 255;
    }
  }
}

void create_xforms(xform *xforms) {
  // 180 degree
  xforms[0].weights[0] = 1.0;
  xforms[0].weights[1] = 0.0;
  xforms[0].weights[2] = 0.0;
  xforms[0].weights[3] = 0.0;
  xforms[0].weight = 33.0f;
  xforms[0].pre_affine = affine{0.5, 0.0, 0.0, 0.0, 0.5, 0.0};
  xforms[0].post_affine = affine{1.0, 0.0, 0.0, 0.0, 1.0, 0.0};
  xforms[0].r = 0.0f;
  xforms[0].g = 1.0f;
  xforms[0].b = 0.0f;
  xforms[0].a = 1.0f;

  xforms[1].weights[0] = 1.0;
  xforms[1].weights[1] = 0.0;
  xforms[1].weights[2] = 0.0;
  xforms[1].weights[3] = 0.0;
  xforms[1].weight = 33.0f;
  xforms[1].pre_affine = affine{0.5, -0.0, 0.5, 0.0, 0.5, 0.0};
  xforms[1].post_affine = affine{1.0, 0.0, 0.0, 0.0, 1.0, 0.0};

  xforms[1].r = 0.0f;
  xforms[1].g = 0.0f;
  xforms[1].b = 1.0f;
  xforms[1].a = 1.0f;

  xforms[2].weights[0] = 1.0;
  xforms[2].weights[1] = 0.0;
  xforms[2].weights[2] = 0.0;
  xforms[2].weights[3] = 0.0;
  xforms[2].weight = 33.0f;
  xforms[2].pre_affine = affine{0.5, 0.0, 0.0, 0.0, 0.5, 0.5};
  xforms[2].post_affine = affine{1.0, 0.0, 0.0, 0.0, 1.0, 0.0};
  xforms[2].r = 1.0f;
  xforms[2].g = 0.0f;
  xforms[2].b = 0.0f;
  xforms[2].a = 1.0f;
}

int main(void) {
  const int num_particles = 100000;
  const int w = 1024;
  const int h = 1024;
  const int num_threads = 1024;
  const int num_blocks = 100;

  xform *xforms;
  pixel *pixels;
  hist *histogram;

  // xforms
  hipMallocManaged(&xforms, NUM_XFORMS * sizeof(xform));

  // Setup number of xforms initially.
  create_xforms(xforms);
  hipDeviceSynchronize();

  // Allocate Unified Memory – accessible from CPU or GPU}
  hipMallocManaged(&pixels, w * h * sizeof(pixel));
  hipMallocManaged(&histogram, w * h * sizeof(hist));

  // Run kernel on 1M elements on the GPU
  sheep<<<num_blocks, num_threads>>>(num_particles, 1000000, histogram, w, h,
                                     xforms);

  histogram_to_image<<<num_blocks, num_threads>>>(histogram, pixels, w, h);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  stbi_write_png("test2.png", w, h, 3, pixels, 0);

  // Free memory
  hipFree(xforms);
  hipFree(pixels);
  hipFree(histogram);

  std::cout << "finished..." << std::endl;

  return 0;
}
